/********************************************************
*
*   This experiment optimizes packet classification
*   in the following aspects:
*     1. Thread assignment
*     2. Memory coalescing
*	
*	Experiment Assumptions:
*		1. 510 Non-overlapping intervals
*		2. 1024 Rules (510 * 1024 element BVs)
*		3. Number of packets varies, 1 kernel
*		4. All packets are already on CPU memory
*		5. All fields needs prefix/range match
*
********************************************************/



#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hipblas.h>

#define FIELD 6
#define RULE 31
#define ALLRULE 128
#define WSIZE 32
#define int_count ALLRULE/sizeof(long int)

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

void header_gen(int**, int**, int, int);
void tree_gen(int**, int, int);
void bv_gen(long int**, long int*, int);
void data_test(int**, int**, long int**, int*, int, int);

__global__ void packet_classify(int* gpu_tree, int* gpu_headers, long int* gpu_bv, int* gpu_match_result, long int* gpu_merge_result, int packet_num, int block_dim){
	__shared__ int gpu_tree_shared[FIELD*RULE];
	int level = 0;
	while(level * block_dim + threadIdx.x < FIELD * RULE){
		gpu_tree_shared[level * block_dim + threadIdx.x] = gpu_tree[level * block_dim + threadIdx.x];
		level++;
	}
	__syncthreads();

	//int index = blockDim.x * blockIdx.x + threadIdx.x;
	//int tree_idx = index / packet_num * RULE;
	int i = 0;
	if (blockDim.x * blockIdx.x + threadIdx.x < packet_num * FIELD){
		while (i < RULE){
			
			i = 2 * i + (gpu_headers[blockDim.x * blockIdx.x + threadIdx.x] <= gpu_tree_shared[(blockDim.x * blockIdx.x + threadIdx.x) / packet_num * RULE+i]) * 1 + (gpu_headers[blockDim.x * blockIdx.x + threadIdx.x] > gpu_tree_shared[(blockDim.x * blockIdx.x + threadIdx.x) / packet_num * RULE+i]) * 2;
		}
		gpu_match_result[blockDim.x * blockIdx.x + threadIdx.x] = i - RULE;
	}

	__syncthreads();
	
	if (blockDim.x * blockIdx.x + threadIdx.x < packet_num * int_count){
		int index = blockDim.x * blockIdx.x + threadIdx.x;
		int packetIdx = index/int_count;
		gpu_merge_result[index] = gpu_bv[gpu_match_result[packetIdx*FIELD]*int_count + index%int_count] &
								  gpu_bv[gpu_match_result[packetIdx*FIELD+1]*int_count + index%int_count];
	/*						  gpu_bv[gpu_match_result[packetIdx*15+2]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+3]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+4]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+5]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+6]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+7]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+8]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+9]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+10]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+11]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+12]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+13]*int_count + index%int_count] &
							  gpu_bv[gpu_match_result[packetIdx*15+14]*int_count + index%int_count];
*/
	}
};


int main(int argc, char** argv){
	if(argc!=4){
		cout<<"usage ./openflow  *Packet_num   *Grid_dim   *Block_dim"<<endl; 
		return 0;
	}
	int packet_num = atoi(argv[1]);
	int grid_dim = atoi(argv[2]);
	int block_dim = atoi(argv[3]);
//	if (grid_dim*block_dim != packet_num*FIELD){
//		cout<<"ERROR: Total number of threads must equal packet_num * FIELD"<<endl;
//		return 1;
//	}
	cout<<"grid_dim: "<<grid_dim<<", block_dim: "<<block_dim<<", packet_num: "<<packet_num<<endl;
cout<<"============================ Experiment Starts ============================"<<endl;
/********************************************************
*	Preparing Data:
*		1. Generate random headers 
*		2. Generate BVs 
*		3. Generate random packets
*		4. Deliberately make some rule-matching packets
********************************************************/
	srand(time(NULL));
	int** tree = new int*[FIELD];
		for(int i = 0; i < FIELD; i++){
			tree[i] = new int[RULE];
		}
	int** headers = new int*[FIELD];
		for (int i = 0; i < FIELD; i++){
			headers[i] = new int[packet_num];
		}
	long int** bv = new long int*[FIELD*(RULE+1)];
		for(int i = 0; i < FIELD*(RULE+1); i++){
			bv[i] = new long int[ALLRULE / sizeof(long int)];
		}
	long int* bv_final = new long int[packet_num * int_count];
	int* match_result = new int[packet_num * FIELD];
	long int* merge_results = new long int[int_count*packet_num];

	tree_gen(tree, FIELD, RULE);
	header_gen(headers, tree, FIELD, packet_num);
	bv_gen(bv, bv_final, packet_num);
	
	//data_test(tree, headers, bv, bv_final, packet_num, 3);

/********************************************************
*	Flatten All the 2D Arrays
********************************************************/
	int* tree_flatten = new int[RULE*FIELD];
	int* headers_flatten = new int[packet_num*FIELD];
	long int* bv_flatten = new long int[FIELD*(RULE+1) * ALLRULE / sizeof(long int)];

	for (int i = 0; i < FIELD; i++){
		for (int j = 0; j < RULE; j++){
			tree_flatten[i*RULE+j] = tree[i][j];
		}
	}
	for (int i = 0; i < FIELD; i++){
		for (int j = 0; j < packet_num; j++){
			headers_flatten[i*packet_num + j] = headers[i][j];
		}
	}
	for (int i = 0; i < FIELD*(RULE+1); i++){
		for (int j = 0; j < ALLRULE / sizeof(long int); j++){
			bv_flatten[i*ALLRULE / sizeof(long int) + j] = bv[i][j];
		}
	}
/********************************************************
*	Declare cuda events for statistical purposes:
*		1. time_memcpyH2D
*		2. time_memcpyD2H
*		3. time_pc
********************************************************/
	float time1, time2, time3;
	hipEvent_t time_memcpyH2D_start, time_memcpyH2D_stop, time_memcpyD2H_start, time_memcpyD2H_stop, time_comp_start, time_comp_stop;
	hipEventCreate(&time_memcpyH2D_start);
	hipEventCreate(&time_memcpyH2D_stop);
	hipEventCreate(&time_memcpyD2H_start);
	hipEventCreate(&time_memcpyD2H_stop);
	hipEventCreate(&time_comp_start);
	hipEventCreate(&time_comp_stop);


/********************************************************
*	Allocate Space in Device:
*		1. gpu_tree 
*		2. gpu_bv 
*		3. gpu_bv_final
*		4. gpu_headers
********************************************************/
	dim3 dimGrid(grid_dim,1);
	dim3 dimBlock(block_dim,1);
	int* gpu_tree;
	int* gpu_headers;
	int* gpu_match_result;
	long int* gpu_merge_results;
	long int* gpu_bv;

	hipMalloc((void**)&gpu_tree, sizeof(int*)*size_t(FIELD*RULE));
		cudaCheckErrors("hipMalloc gpu_tree");
	hipMalloc((void**)&gpu_headers, sizeof(int)*FIELD*packet_num);
		cudaCheckErrors("hipMalloc gpu_headers");
	hipMalloc((void**)&gpu_bv, sizeof(long int) * FIELD*(RULE+1) * int_count);
		cudaCheckErrors("hipMalloc gpu_bv");
	hipMalloc((void**)&gpu_match_result, sizeof(int)*packet_num*FIELD);
		cudaCheckErrors("hipMalloc gpu_match_result");
	hipMalloc((void**)&gpu_merge_results, sizeof(long int)*packet_num*int_count);
		cudaCheckErrors("hipMalloc gpu_merge_results");

	hipEventRecord(time_memcpyH2D_start, 0);
	
	hipMemcpy(gpu_tree, tree_flatten, sizeof(int)*RULE*FIELD, hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy gpu_tree");
	hipMemcpy(gpu_headers, headers_flatten, sizeof(int)*FIELD*packet_num, hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy gpu_headers");
	hipMemcpy(gpu_bv, bv_flatten, sizeof(long int) * FIELD*(RULE+1) * int_count, hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy gpu_bv");

	hipEventRecord(time_memcpyH2D_stop, 0);
	hipEventSynchronize(time_memcpyH2D_stop);
	hipEventElapsedTime(&time1, time_memcpyH2D_start, time_memcpyH2D_stop);
	hipEventDestroy(time_memcpyH2D_stop);
	hipEventDestroy(time_memcpyH2D_start);

	cout<<endl<<"*	1. Time for memcpy H2D: "<<time1<<"ms, Total bytes copied: "<<endl;
	cout<<"    		-> Tree: "<< sizeof(int)*RULE*FIELD<<" Bytes"<<endl;
	cout<<"    		-> Headers: "<< sizeof(long int)*FIELD*packet_num<<" Bytes"<<endl;
	cout<<"    		-> Bv: "<< sizeof(long int) * FIELD*(RULE+1) * int_count<<" Bytes"<<endl;
	cout<<"    		-> Total Memory Copy: "<< sizeof(int)*RULE*FIELD + sizeof(long int)*FIELD*packet_num + sizeof(long int) * FIELD*(RULE+1) * int_count<<" Bytes"<<endl;



/********************************************************
*	Main Packet Classification Process:
*		1. Function Call
*		2. Timing
*		3. Memory copy back (gpu_bv_final)
********************************************************/

	hipEventRecord(time_comp_start, 0);

	packet_classify<<<dimGrid, dimBlock>>>(gpu_tree, gpu_headers, gpu_bv, gpu_match_result, gpu_merge_results, packet_num, block_dim);
	cudaCheckErrors("Kernel fail");

	hipEventRecord(time_comp_stop, 0);
	hipEventSynchronize(time_comp_stop);
	hipEventElapsedTime(&time2, time_comp_start, time_comp_stop);
	hipEventDestroy(time_comp_stop);
	hipEventDestroy(time_comp_start);
	cout<<endl<<"*	2. Time for GPU computation: "<<time2<<"ms, GPU throughput: "<<packet_num/time2/1000<<" MPPS"<<endl;


	hipEventRecord(time_memcpyD2H_start, 0);
	
	hipMemcpy(bv_final, gpu_merge_results, sizeof(long int) * packet_num * int_count, hipMemcpyDeviceToHost);
	
	hipEventRecord(time_memcpyD2H_stop, 0);
	hipEventSynchronize(time_memcpyD2H_stop);
	hipEventElapsedTime(&time3, time_memcpyD2H_start, time_memcpyD2H_stop);
	hipEventDestroy(time_memcpyD2H_stop);
	hipEventDestroy(time_memcpyD2H_start);
	cout<<endl<<"*	3. Time for memcpy H2D: "<<time3<<"ms, Total bytes copied: "<<endl;
	cout<<"    		-> Bv_final: "<< sizeof(long int) * packet_num * int_count<<" Bytes"<<endl<<endl;

	cout<<endl<<">>>>>> Total GPU throughput: "<<packet_num/(time1 + time2 + time3)/1000<<" MPPS"<<endl;

	//data_test(tree, headers, bv, bv_final, packet_num, 8);

/********************************************************
*	Clear Memory:
*		1. Dynamic allocations on host
*		2. cudaFrees
********************************************************/
	hipFree(gpu_tree);
	cudaCheckErrors("Free gpu_tree fail");
	hipFree(gpu_bv);
	cudaCheckErrors("Free bv fail");
	hipFree(gpu_headers);
	cudaCheckErrors("Free gpu_headers fail");
	hipFree(gpu_match_result);
	cudaCheckErrors("Free gpu_match_result fail");
	hipFree(gpu_merge_results);
	cudaCheckErrors("Free gpu_merge_results fail");

	for (int i = 0; i < FIELD; i++){
		delete tree[i];
	}
	for(int i = 0; i < FIELD; i++){
		delete headers[i];
	}
	for(int i = 0; i < FIELD*(RULE+1); i++){
		delete bv[i];
	}
	delete tree;
	delete bv;
	delete headers;
	delete bv_final;
	delete match_result;
	delete tree_flatten;
	delete headers_flatten;
	delete bv_flatten;
	delete merge_results;
    cout<<"============================ Experiment Ends ============================"<<endl;
	return 0;
}




void tree_gen(int** tree, int field, int rule){
	for(int i = 0; i < field; i++){
		tree[i][0] = rand() % 100;
		int temp[rule];
		temp[0] = tree[i][0];
		for (int j = 1; j < rule; j++){
			temp[j] = temp[j-1] + rand() % 20 + 1;
		}
		int temp_index = rule-1, tree_index = rule -1, level = log(rule+1) / log(2);
		int step_index = level;
		while (step_index >= 1){
			int step = pow(2, (level - step_index + 1));
			while (temp_index >= 0){
				tree[i][tree_index] = temp[temp_index];
				temp_index -= step;
				tree_index--;
			}
			step_index--;
			temp_index = rule - 1 - (pow(2, level - step_index) - 1);
		}
	}
}
void header_gen(int** headers, int** tree, int field, int packet_num){
	for (int i = 0; i < field; i++){
		for(int j = 0; j < packet_num; j++){
			headers[i][j] = rand() % 6000;
		}
	
	}
}
void bv_gen(long int ** bv, long int* bv_final, int packet_num){
	for (int i = 0; i < ALLRULE / sizeof(long int); i++){
		for (int j = 0; j < FIELD*(RULE+1); j++){
			bv[j][i] = rand() % 1000000;
		}
	}
	for(int i = 0; i < packet_num; i++){
		bv_final[i] = -1;
	}
}
void data_test(int** tree, int** headers, long int** bv, int* bv_final, int packet_num, int type){
	if (type > 15 | type == 0){
		return;
	}
	if (type % 2 == 1){
		cout<<"Tree: "<<endl;
		for(int i = 0; i < RULE; i++){
			cout<<"Line: "<<i<<": ";
			for(int j = 0; j < FIELD; j++){
				cout<<tree[j][i]<<" ";
			}
			cout<<endl;
		}
	}
	if (type % 4 == 2 | type % 4 == 3){
		cout<<endl<<"Headers: "<<endl;
		for(int i = 0; i < packet_num; i++){
			cout<<"Header "<<i<<": ";
			for(int j = 0; j < FIELD; j++){
				cout<<headers[j][i]<<" ";
			}
			cout<<endl;
		}
	}
	if (type % 8 == 4 | type % 8 == 5 | type % 8 == 6 | type % 8 == 7){
		cout<<endl<<"bv: "<<endl;
		for(int i = 0; i < ALLRULE; i++){
			cout<<"Line "<<i<<": ";
			for (int j = 0; j < FIELD*(RULE+1); j++){
			cout<<bv[j][i]<<" ";
			}
			cout<<endl;
		}
	}

	if (type > 7){
		cout<<endl<<"bv_final: "<<endl;
		for(int i = 0; i < packet_num; i++){
			cout<<bv_final[i]<<" ";
		}
		cout<<endl;
	}
	cout<<"============== End of Print =============="<<endl;
}



