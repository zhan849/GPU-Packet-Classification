#include <iostream>
#include <pthread.h>
#include <sys/types.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hipblas.h>
#include "gpu_pc_v2_func.h"

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;


int main(int argc, char** argv){
    if(argc!=4){
        cout<<"usage ./openflow  *Packet_num   *Grid_dim   *Block_dim   *Grid_dim_merge   *Block_dim_merge"<<endl; 
        return 0;
    }
    int packet_num = atoi(argv[1]);
    int grid_dim = atoi(argv[2]);
    int block_dim = atoi(argv[3]);
    if (grid_dim*block_dim != packet_num*FIELD){
        cout<<"ERROR: Total number of threads in stage 1 must equal packet_num * FIELD"<<endl;
        return 1;
    }
    cout<<"============================ Experiment Starts ============================"<<endl;
/********************************************************
*   Declare data structs
********************************************************/
    Stage1_Data S1;
    Stage2_Data S2;
    S1.packet_num = packet_num;
    S1.block_dim = block_dim;
/********************************************************
*   Preparing Data:
*       1. Generate random header
*       2. Generate BVs 
*       3. Generate random packets
*       4. Deliberately make some rule-matching packets
********************************************************/
    srand(time(NULL));
    int** tree = new int*[FIELD];
        for(int i = 0; i < FIELD; i++){
            tree[i] = new int[RULE];
        }
    int** header = new int*[FIELD];
        for (int i = 0; i < FIELD; i++){
            header[i] = new int[packet_num];
        }
    long int** bv = new long int*[FIELD*(RULE+1)];
        for(int i = 0; i < FIELD*(RULE+1); i++){
            bv[i] = new long int[int_count];
        }
    S2.bv_final = new long int[packet_num*int_count];
    S2.match_result = new int[packet_num * FIELD];
    S2.merge_result_partial = new long int[int_count*packet_num];

    tree_gen(tree, FIELD, RULE);
    header_gen(header, tree, FIELD, packet_num);
    bv_gen(bv, S2.bv_final, packet_num);

/********************************************************
*   Flatten All the 2D Arrays
********************************************************/
    S1.tree = new int[RULE*FIELD];
    S1.header = new int[packet_num*FIELD];
    S2.bv = new long int[FIELD*(RULE+1) * int_count];

    for (int i = 0; i < FIELD; i++){
        for (int j = 0; j < RULE; j++){
            S1.tree[i*RULE+j] = tree[i][j];
        }
    }
    for (int i = 0; i < FIELD; i++){
        for (int j = 0; j < packet_num; j++){
            S1.header[i*packet_num + j] = header[i][j];
        }
    }
    for (int i = 0; i < FIELD*(RULE+1); i++){
        for (int j = 0; j < int_count; j++){
            S2.bv[ i * int_count + j] = bv[i][j];
        }
    }

/********************************************************
*   Setup Timers:
*       1. gpu_tree 
*       2. gpu_header 
*       3. gpu_match_result
********************************************************/
    float time1, time2, time3, time4 = 0;
    hipEvent_t time_search_memcpyH2D_start, time_search_memcpyH2D_stop, 
                time_search_memcpyD2H_start, time_search_memcpyD2H_stop, 
                time_gpu_start, time_gpu_stop;
    struct timespec cpu_merge_start, cpu_merge_stop;

    hipEventCreate(&time_search_memcpyH2D_start);
    hipEventCreate(&time_search_memcpyH2D_stop);
    hipEventCreate(&time_search_memcpyD2H_start);
    hipEventCreate(&time_search_memcpyD2H_stop);
    hipEventCreate(&time_gpu_start);
    hipEventCreate(&time_gpu_stop);

/********************************************************
*   Allocate Space in Device:
*       1. gpu_tree 
*       2. gpu_header 
*       3. gpu_match_result
********************************************************/
    dim3 dimGrid(grid_dim,1);
    dim3 dimBlock(block_dim,1);

    int* gpu_tree;
    int* gpu_header;
    int* gpu_match_result;

    hipMalloc((void**)&gpu_tree, sizeof(int)*FIELD*RULE);
        cudaCheckErrors("hipMalloc gpu_tree");
    hipMalloc((void**)&gpu_header, sizeof(int)*FIELD*packet_num);
        cudaCheckErrors("hipMalloc gpu_headers");
    hipMalloc((void**)&gpu_match_result, sizeof(int)*packet_num*FIELD);
        cudaCheckErrors("hipMalloc gpu_match_result");
    
    hipEventRecord(time_search_memcpyH2D_start, 0);

    hipMemcpy(gpu_tree, S1.tree, sizeof(int)*RULE*FIELD, hipMemcpyHostToDevice);
        cudaCheckErrors("hipMemcpy gpu_tree");
    hipMemcpy(gpu_header, S1.header, sizeof(int)*FIELD*packet_num, hipMemcpyHostToDevice);
        cudaCheckErrors("hipMemcpy gpu_headers");

    hipEventRecord(time_search_memcpyH2D_stop, 0);
    hipEventSynchronize(time_search_memcpyH2D_stop);
    hipEventElapsedTime(&time1, time_search_memcpyH2D_start, time_search_memcpyH2D_stop);
    hipEventDestroy(time_search_memcpyH2D_stop);
    hipEventDestroy(time_search_memcpyH2D_start);
    cout<<endl<<"Number of Packets: "<<packet_num<<endl;
    cout<<endl<<">>>>>>[Stage 1: Search][GPU] "<<"grid_dim: "<<grid_dim<<", block_dim: "<<block_dim<<endl;
    cout<<endl<<"*  1. Time for memcpy H2D: "<<time1<<"ms, Total bytes copied: "<<endl;
    cout<<"         -> Tree: "<< sizeof(int)*RULE*FIELD<<" Bytes"<<endl;
    cout<<"         -> Headers: "<< sizeof(int)*FIELD*packet_num<<" Bytes"<<endl;
    cout<<"         -> Total Memory Copy: "<< sizeof(int)*RULE*FIELD + sizeof(int)*FIELD*packet_num<<" Bytes"<<endl;

/********************************************************
*   Main Packet Classification Process [Search][GPU]
*       1. Function Call
*       2. Timing
*       3. Memory copy back (gpu_bv_final)
********************************************************/
    hipEventRecord(time_gpu_start, 0);

    packet_classify<<<dimGrid, dimBlock>>>(gpu_tree, gpu_header, gpu_match_result, S1.packet_num, S1.block_dim);

    cudaCheckErrors("Search fail");
    hipEventRecord(time_gpu_stop, 0);
    hipEventSynchronize(time_gpu_stop);
    hipEventElapsedTime(&time2, time_gpu_start, time_gpu_stop);
    hipEventDestroy(time_gpu_stop);
    hipEventDestroy(time_gpu_start);
    cout<<endl<<"*  2. Time for GPU computation: "<<time2<<"ms, GPU throughput: "<<packet_num/time2/1000<<" MPPS"<<endl;


    hipEventRecord(time_search_memcpyD2H_start, 0);

    hipMemcpy(S2.match_result, gpu_match_result, sizeof(int)*packet_num*FIELD, hipMemcpyDeviceToHost);

    hipEventRecord(time_search_memcpyD2H_stop, 0);
    hipEventSynchronize(time_search_memcpyD2H_stop);
    hipEventElapsedTime(&time3, time_search_memcpyD2H_start, time_search_memcpyD2H_stop);
    hipEventDestroy(time_search_memcpyD2H_stop);
    hipEventDestroy(time_search_memcpyD2H_start);
    cout<<endl<<"*  3. Time for memcpy D2H: "<<time3<<"ms, Total bytes copied: "<<endl;
    cout<<"         -> Match_result: "<< sizeof(int)*packet_num*FIELD<<" Bytes"<<endl<<endl;

/********************************************************
*   Main Packet Classification Process [Merge][CPU]
********************************************************/

    if (clock_gettime(CLOCK_REALTIME, &cpu_merge_start) == -1){
        perror("clock_gettime: cpu_merge_start");
    } 
    for (int i = 0; i < packet_num; i++){
        for (int j = 0; j < int_count; j++){
            S2.bv_final[i*int_count + j] = bv[S2.match_result[i*FIELD+0]][j] &
                                        bv[S2.match_result[i*FIELD+1]][j] &
                                        bv[S2.match_result[i*FIELD+2]][j] &
                                        bv[S2.match_result[i*FIELD+3]][j] &
                                        bv[S2.match_result[i*FIELD+4]][j] &
                                        bv[S2.match_result[i*FIELD+5]][j] &
                                        bv[S2.match_result[i*FIELD+6]][j] &
                                        bv[S2.match_result[i*FIELD+7]][j] &
                                        bv[S2.match_result[i*FIELD+8]][j] &
                                        bv[S2.match_result[i*FIELD+9]][j] &
                                        bv[S2.match_result[i*FIELD+10]][j] &
                                        bv[S2.match_result[i*FIELD+11]][j] &
                                        bv[S2.match_result[i*FIELD+12]][j] &
                                        bv[S2.match_result[i*FIELD+13]][j] &
                                        bv[S2.match_result[i*FIELD+14]][j]; 
        }
    }
    if (clock_gettime(CLOCK_REALTIME, &cpu_merge_stop) == -1){
        perror("clock_gettime: cpu_merge_stop");
    }
    
    time4 += (cpu_merge_stop.tv_sec - cpu_merge_start.tv_sec) * 1000 + 
             (cpu_merge_stop.tv_nsec - cpu_merge_start.tv_nsec) / 1e6;
    cout<<endl<<">>>>>>[Stage 2: Merge][CPU] "<<endl;
    cout<<endl<<"*  1. Time for Merge: "<<time4<<"ms"<<endl;
    cout<<endl<<"*  2. CPU throughput: "<<packet_num/(time4)/1000<<" MPPS"<<endl;
    cout<<endl<<">>>>>> Total throughput: "<<packet_num/(time1 + time2 + time3 + time4)/1000<<" MPPS"<<endl;
/********************************************************
*   Clear Memory:
*       1. Dynamic allocations on host
*       2. cudaFrees
********************************************************/
    hipFree(gpu_tree);
    cudaCheckErrors("Free gpu_tree fail");
    hipFree(gpu_header);
    cudaCheckErrors("Free gpu_headers fail");
    hipFree(gpu_match_result);
    cudaCheckErrors("Free gpu_match_result fail");

    for (int i = 0; i < FIELD; i++){
        delete tree[i];
    }
    for(int i = 0; i < FIELD; i++){
        delete header[i];
    }
    for(int i = 0; i < FIELD*(RULE+1); i++){
        delete bv[i];
    }
    
    delete tree;
    delete bv;
    delete header;
    
    delete S2.bv_final;
    delete S2.match_result;
    delete S1.tree;
    delete S1.header;
    delete S2.bv;
    delete S2.merge_result_partial;
    //delete partial_merge_threads;
    //delete final_merge_threads;

     cout<<"============================ Experiment Ends ============================"<<endl;
    return 0;
}